#include "hip/hip_runtime.h"
#include "complex.h"
#include "mandelbrot.h"

#define BLOCK_SIZE	16
#define X(x)	minX+(maxX-minX)/resolution*x
#define Y(y)	minY+(maxY-minY)/resolution*y

__global__ void gpu_mandelbrot(unsigned *uvm_mandelbrot) {
	// get x and y values of thread
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	// get index of device memory by x and y values
	int idx = y * resolution + x;

	if(x < resolution && y < resolution) {
		// complex number on complex plain with (-2.2, -1.5), (1.0, 1.6)
		Complex<float> c(X(x), Y(y));
		// complex number Zn for find iteration count, and Z0 = c
		Complex<float> z = c;

		// iteration count 
		int itr_cnt = 0;

		// find iteration count
		for(itr_cnt = 0; itr_cnt < max_iterations; itr_cnt++) {
			// if magnitude of Z^2 overs escape(=4.0), exit loop and get iteration cnt
			if(z.magnitude2() > escape)
				break;
			// find Zn by Z = Z^2 + C
			z = z * z + c;
		}
		// set iteraction count number to device memory with idx
		uvm_mandelbrot[idx] = itr_cnt;	
	}
}

void calc_mandelbrot(void) {
    	/* Assignment 5: GPU Programming */
	// Copies for 'mandelbrot'
	unsigned* uvm_mandelbrot;

	// Allocate Device Memory
	size_t display_size = resolution * resolution * sizeof(unsigned);
	hipMalloc(&uvm_mandelbrot, display_size);

	// Define threads and thread blocks
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 numBlocks(resolution / BLOCK_SIZE, resolution / BLOCK_SIZE);

	// Launch Cuda Kernel 
	gpu_mandelbrot<<<numBlocks, threadsPerBlock>>>(uvm_mandelbrot);

	// Wait for Device to complete kernel task
	hipDeviceSynchronize();

	// Copy the result of kernel
	hipMemcpy(mandelbrot, uvm_mandelbrot, display_size, hipMemcpyDeviceToHost);

	// Free Device Memory
	hipFree(uvm_mandelbrot);
}

